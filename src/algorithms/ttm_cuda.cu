#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI/algorithm.hpp>
#include <cassert>
#include <memory>
#include <utility>
#include <vector>
#include <ParTI/error.hpp>
#include <ParTI/errcode.hpp>
#include <ParTI/sptensor.hpp>
#include <ParTI/tensor.hpp>
#include <ParTI/timer.hpp>

namespace pti {

namespace {


void __global__ ttm_cuda_kernel(
    size_t const *__restrict__ fiberidx, size_t const *__restrict__ X_indices_m,
    size_t nrows, size_t ncols, size_t Y_chunk_size, size_t Y_subchunk_size, size_t X_chunk_size, size_t U_stride,
    Scalar *__restrict__ Y_values, Scalar const *__restrict__ X_values, Scalar const *__restrict__ U_values
) {
    size_t i = blockIdx.x;             // i := mode-n fiber
    size_t inz_begin = fiberidx[i];    // inz_begin/end := global indices for monde-n fiber of X
    size_t inz_end = fiberidx[i + 1];
    size_t r = threadIdx.x;
    for(size_t k = threadIdx.y; k < Y_subchunk_size; k += blockDim.y) {
        Scalar accumulate = 0;
        for(size_t j = inz_begin; j < inz_end; ++j) { // loop over fiber i
            size_t c = X_indices_m[j]; // get mode-n index of X: c ∈ [1, size(mode-n)]
            if(r < nrows && c < ncols) {
                accumulate += X_values[j * X_chunk_size + k] * U_values[r * U_stride + c];
            }
        }
        Y_values[i * Y_chunk_size + r * Y_subchunk_size + k] += accumulate;
    }
}

/* impl_num = 15 */
__global__ void spt_TTMRankRBNnzKernelSM(
    Scalar *Y_val, 
    size_t Y_stride, size_t Y_nnz,
    const Scalar * __restrict__ X_val, 
    size_t X_nnz, 
    const size_t * __restrict__ X_inds_m,
    const size_t * __restrict__ fiberidx_val, 
    const Scalar * __restrict__ U_val, 
    size_t U_nrows, 
    size_t U_ncols, 
    size_t U_stride) 
{
    extern __shared__ Scalar mem_pool[];
    Scalar * const Y_shr = (Scalar *) mem_pool; // size U_ncols

    size_t num_loops_nnz = 1;
    size_t const nnz_per_loop = gridDim.x * blockDim.y;
    if(Y_nnz > nnz_per_loop) {
        num_loops_nnz = (Y_nnz + nnz_per_loop - 1) / nnz_per_loop;
    }
     
    // Global indices of Y: Fiber = x and Inner fiber = r
    // Local indices: tidx and tidy
    const size_t tidx = threadIdx.x;
    const size_t tidy = threadIdx.y;
    size_t x;
    const size_t num_loops_r = U_ncols / blockDim.x;
    const size_t rest_loop = U_ncols - num_loops_r * blockDim.x;
    size_t r; // column idx of U


    for(size_t l=0; l<num_loops_r; ++l) { // blockDim.x parallelised over cols(U)
        r = tidx + l * blockDim.x; // r: column idx of U
        for(size_t nl=0; nl<num_loops_nnz; ++nl) { // Grid strided-pattern?
            x = blockIdx.x * blockDim.y + tidy + nl * nnz_per_loop;

            Y_shr[tidy * blockDim.x + tidx] = 0;
            __syncthreads();

            if(x < Y_nnz) { // Why is this not above at line 348
                const size_t inz_begin = fiberidx_val[x];
                const size_t inz_end = fiberidx_val[x+1];
                for(size_t i = inz_begin; i < inz_end; ++i) { // loop over a n-fiber
                    const size_t row = X_inds_m[i]; // row of U
                    // Loop over nnz in n-fiber of X and multiply with corresponding
                    // U col elements and accumulate in single element of Y
                    Y_shr[tidy*blockDim.x + tidx] += X_val[i] * U_val[row*U_stride + r];  // Original
                    // Y_shr[tidy*blockDim.x + tidx] += X_val[i] * U_val[r*U_stride + row]; 
                }
                __syncthreads();

                Y_val[x*Y_stride + r] = Y_shr[tidy*blockDim.x + tidx];
                __syncthreads();
            }
        }
    }


    if(rest_loop > 0 && tidx < rest_loop) {
        r = tidx + num_loops_r * blockDim.x;

        for(size_t nl=0; nl<num_loops_nnz; ++nl) {
            x = blockIdx.x * blockDim.y + tidy + nl * nnz_per_loop;

            Y_shr[tidy * blockDim.x + tidx] = 0;
            __syncthreads();

            if(x < Y_nnz) {
                const size_t inz_begin = fiberidx_val[x];
                const size_t inz_end = fiberidx_val[x+1];
                for(size_t i = inz_begin; i < inz_end; ++i) {
                    const size_t row = X_inds_m[i];
                    Y_shr[tidy*blockDim.x + tidx] += X_val[i] * U_val[row*U_stride + r];  // Original
                    // Y_shr[tidy*blockDim.x + tidx] += X_val[i] * U_val[r*U_stride + row]; 
                }
                __syncthreads();

                Y_val[x*Y_stride + r] = Y_shr[tidy*blockDim.x + tidx];
                __syncthreads();
            }
        }
    }

}

}


SparseTensor tensor_times_matrix_cuda(SparseTensor& X, Tensor& U, size_t mode, CudaDevice* cuda_dev, bool skip_sort) {
    size_t nmodes = X.nmodes;
    size_t nspmodes = X.sparse_order.size();

    ptiCheckError(mode >= nmodes, ERR_SHAPE_MISMATCH, "mode >= X.nmodes");
    ptiCheckError(X.is_dense(cpu)[mode], ERR_UNKNOWN, "X.is_dense[mode] != false");

    ptiCheckError(U.nmodes != 2, ERR_SHAPE_MISMATCH, "U.nmodes != 2");
    ptiCheckError(U.storage_order(cpu)[0] != 0, ERR_SHAPE_MISMATCH, "U.storage_order[0] != 0");
    ptiCheckError(U.storage_order(cpu)[1] != 1, ERR_SHAPE_MISMATCH, "U.storage_order[1] != 1");

    size_t nrows = U.shape(cpu)[0];
    size_t ncols = U.shape(cpu)[1];
    size_t Ustride = U.strides(cpu)[1];

    ptiCheckError(X.shape(cpu)[mode] != nrows, ERR_SHAPE_MISMATCH, "X.shape[mode] != U.ncols");

    if(skip_sort) {
        ptiCheckError(X.sparse_order(cpu)[nspmodes - 1] != mode, ERR_SHAPE_MISMATCH, "X.sparse_order[-1] != mode");
    } else {
        Timer timer_sort(cpu);
        timer_sort.start();

        std::unique_ptr<size_t[]> sort_order(new size_t [nspmodes]);
        for(size_t m = 0, i = 0; m < nspmodes; ++m) {
            size_t sort_order_mode = X.sparse_order(cpu)[m];
            if(sort_order_mode != mode) {
                sort_order[i] = sort_order_mode;
                ++i;
            }
        }
        sort_order[nspmodes - 1] = mode;
        X.sort_index(sort_order.get());

        timer_sort.stop();
        timer_sort.print_elapsed_time("CUDA TTM Sort");
    }
    std::printf("X = %s\n", X.to_string(1, 10).c_str());

    std::unique_ptr<size_t[]> Y_shape(new size_t [nmodes]);
    for(size_t m = 0; m < nmodes; ++m) {
        if(m != mode) {
            Y_shape[m] = X.shape(cpu)[m];
        } else {
            Y_shape[m] = nrows;
        }
    }
    bool const* X_is_dense = X.is_dense(cpu);
    std::unique_ptr<bool[]> Y_is_dense(new bool [nmodes]);
    for(size_t m = 0; m < nmodes; ++m) {
        Y_is_dense[m] = X_is_dense[m] || m == mode;
    }

    SparseTensor Y(nmodes, Y_shape.get(), Y_is_dense.get());
    size_t* X_dense_order = X.dense_order(cpu);
    size_t* Y_dense_order = Y.dense_order(cpu);
    for(size_t m = 0; m < Y.dense_order.size() - 1; ++m) {
        Y_dense_order[m] = X_dense_order[m];
    }
    Y_dense_order[Y.dense_order.size() - 1] = mode;
    Y.sort_index(X.sparse_order(cpu));

    Timer timer_setidx(cpu);
    timer_setidx.start();

    std::vector<size_t> fiberidx;
    set_semisparse_indices_by_sparse_ref(Y, fiberidx, X, mode);

    timer_setidx.stop();
    timer_setidx.print_elapsed_time("CUDA TTM SetIdx");
    printf("Fiberidx length = %zu\n", fiberidx.size());

    Scalar* X_values = X.values(cuda_dev->mem_node);
    printf("X_values\n");
    Scalar* Y_values = Y.values(cuda_dev->mem_node);
    printf("Y_values\n");
    Scalar* U_values = U.values(cuda_dev->mem_node);
    printf("U_values\n");
    size_t* X_indices_m = X.indices[mode](cuda_dev->mem_node);
    printf("Allocating for fiberidx: %f GB\n", fiberidx.size() * sizeof(size_t) * 1e-9);
    size_t *dev_fiberidx = (size_t *) session.mem_nodes[cuda_dev->mem_node]->malloc(fiberidx.size() * sizeof (size_t));
    printf("Allocated\n");
    session.mem_nodes[cuda_dev->mem_node]->memcpy_from(dev_fiberidx, fiberidx.data(), *session.mem_nodes[cpu], fiberidx.size() * sizeof (size_t));

    size_t Y_subchunk_size = X.chunk_size;
    size_t Y_num_subchunks = Y.strides(cpu)[mode];
    assert(Y_num_subchunks * Y_subchunk_size == Y.chunk_size);

    const size_t max_nblocks = 32768;
    const size_t max_nthreads_per_block = 256;
    size_t max_nthreadsy = 32;

    size_t nthreadsx = 1;
    size_t nthreadsy = 1;
    size_t all_nblocks = 0;
    size_t nblocks = 0;
    size_t shmen_size = 0;

    if(ncols <= max_nthreadsy)
        nthreadsx = ncols;
    else
        nthreadsx = max_nthreadsy;
    nthreadsy = max_nthreads_per_block / nthreadsx;

    size_t Y_nnz = Y.num_chunks;
    if(Y_nnz < nthreadsy) {
        nthreadsy = Y_nnz;
        nblocks = 1;
    } else {
        all_nblocks = (Y_nnz + nthreadsy -1) / nthreadsy;
        if(all_nblocks < max_nblocks) {
            nblocks = all_nblocks;
        } else {
            nblocks = max_nblocks;
        }
    }
    shmen_size = nthreadsx * nthreadsy * sizeof(Scalar);
    assert(shmen_size >= nthreadsx * nthreadsy * sizeof (Scalar));
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("SHMEM size: %lu  (%lu bytes)\n", shmen_size / sizeof(Scalar), shmen_size);
    printf("all_nblocks: %lu, nthreadsx: %lu, nthreadsy: %lu\n", all_nblocks, nthreadsx, nthreadsy);

    printf("X_nnz: %lu\n", X.num_chunks);
    printf("U_rows: %lu; U_cols: %lu; U_stride: %lu\n", nrows, ncols, Ustride);
    printf("Y_nnz: %lu; Y_stride: %lu\n", Y.num_chunks, Y.chunk_size);




    Timer timer_kernel(cuda_dev->device_id);
    timer_kernel.start();
    size_t kernel_blockDim_y = std::min(Y_subchunk_size, 1024 / Y_num_subchunks);
    assert(kernel_blockDim_y > 0);
    // std::fprintf(stderr, "[CUDA TTM Kernel] Launch ttm_cuda_kernel<<<%zu, (%zu, %zu), 0>>()\n", Y.num_chunks, Y_num_subchunks, kernel_blockDim_y);
    // ttm_cuda_kernel<<<Y.num_chunks, dim3(Y_num_subchunks, kernel_blockDim_y), 0>>>(dev_fiberidx, X_indices_m, nrows, ncols, Y.chunk_size, Y_subchunk_size, X.chunk_size, Ustride, Y_values, X_values, U_values);

//     printf("[CUDA SpTns * Mtx] spt_TTMRankRBNnzKernelSM<<<%lu, (%lu, %lu), %lu>>>\n", nblocks, nthreadsx, nthreadsy, shmen_size);
// spt_TTMRankRBNnzKernelSM<<<nblocks, dimBlock, shmen_size>>>(
//     dev_fiberidx, X_indices_m, nrows, ncols, Ustride,
//     Y.chunk_size, Y.num_chunks, X.num_chunks, X_values, U_values, Y_values);

    printf("[CUDA SpTns * Mtx] spt_TTMRankRBNnzKernelSM<<<%lu, (%lu, %lu), %lu>>>\n", nblocks, nthreadsx, nthreadsy, shmen_size);
    spt_TTMRankRBNnzKernelSM<<<nblocks, dimBlock, shmen_size>>>(
        Y_values, Y.chunk_size, Y.num_chunks,
        X_values, X.num_chunks, X_indices_m,
        dev_fiberidx, U_values, nrows, ncols, Ustride);

    int result = hipDeviceSynchronize();
    timer_kernel.stop();
    timer_kernel.print_elapsed_time("CUDA TTM Kernel");
    ptiCheckCUDAError(result != 0);

    session.mem_nodes[cuda_dev->mem_node]->free(dev_fiberidx);

    return Y;
}

}
